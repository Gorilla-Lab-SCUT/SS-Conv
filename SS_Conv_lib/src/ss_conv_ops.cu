/*
--------------------------------------------------------
Sparse Steerable Convolution Lib.

Template Specialzations for Different Data Types. 
Written by Hongyang Li and Jiehong Lin
Modified https://github.com/dvlab-research/PointGroup
--------------------------------------------------------
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "indice_maxpool.h"
#include "indice_avgpool.h"
#include "indice_conv.h"
#include "voxelize.h"


std::vector<torch::Tensor> get_indice_pairs_2d(torch::Tensor indices, int64_t batchSize,
        std::vector<int64_t> outSpatialShape, std::vector<int64_t> spatialShape,
        std::vector<int64_t> kernelSize, std::vector<int64_t> stride,
        std::vector<int64_t> padding, std::vector<int64_t> dilation,
        std::vector<int64_t> outPadding, int64_t _subM, int64_t _transpose){
    return getIndicePair<2>(indices, batchSize,
        outSpatialShape, spatialShape,
        kernelSize, stride,
        padding, dilation,
        outPadding, _subM, _transpose);
}
std::vector<torch::Tensor> get_indice_pairs_3d(torch::Tensor indices, int64_t batchSize,
        std::vector<int64_t> outSpatialShape, std::vector<int64_t> spatialShape,
        std::vector<int64_t> kernelSize, std::vector<int64_t> stride,
        std::vector<int64_t> padding, std::vector<int64_t> dilation,
        std::vector<int64_t> outPadding, int64_t _subM, int64_t _transpose){
    return getIndicePair<3>(indices, batchSize,
        outSpatialShape, spatialShape,
        kernelSize, stride,
        padding, dilation,
        outPadding, _subM, _transpose);
}
std::vector<torch::Tensor> get_indice_pairs_grid_2d(torch::Tensor indices, torch::Tensor gridOut, int64_t batchSize,
        std::vector<int64_t> outSpatialShape, std::vector<int64_t> spatialShape,
        std::vector<int64_t> kernelSize, std::vector<int64_t> stride,
        std::vector<int64_t> padding, std::vector<int64_t> dilation,
        std::vector<int64_t> outPadding, int64_t _subM, int64_t _transpose){
    return getIndicePairPreGrid<2>(indices, gridOut, batchSize,
        outSpatialShape, spatialShape,
        kernelSize, stride,
        padding, dilation,
        outPadding, _subM, _transpose);
}
std::vector<torch::Tensor> get_indice_pairs_grid_3d(torch::Tensor indices, torch::Tensor gridOut, int64_t batchSize,
        std::vector<int64_t> outSpatialShape, std::vector<int64_t> spatialShape,
        std::vector<int64_t> kernelSize, std::vector<int64_t> stride,
        std::vector<int64_t> padding, std::vector<int64_t> dilation,
        std::vector<int64_t> outPadding, int64_t _subM, int64_t _transpose){
    return getIndicePairPreGrid<3>(indices, gridOut, batchSize,
        outSpatialShape, spatialShape,
        kernelSize, stride,
        padding, dilation,
        outPadding, _subM, _transpose);
}
torch::Tensor indice_conv_fp32(torch::Tensor features, torch::Tensor filters,
                       torch::Tensor indicePairs, torch::Tensor indiceNum,
                       int64_t numActOut, int64_t _inverse, int64_t _subM){
    return indiceConv<float>(features, filters, indicePairs, indiceNum, numActOut, _inverse, _subM);
}
std::vector<torch::Tensor> indice_conv_backward_fp32(torch::Tensor features, torch::Tensor filters,
                 torch::Tensor outGrad, torch::Tensor indicePairs, torch::Tensor indiceNum,
                 int64_t _inverse, int64_t _subM){
    return indiceConvBackward<float>(features, filters, outGrad, indicePairs, indiceNum, _inverse, _subM);
}
torch::Tensor indice_conv_half(torch::Tensor features, torch::Tensor filters,
                       torch::Tensor indicePairs, torch::Tensor indiceNum,
                       int64_t numActOut, int64_t _inverse, int64_t _subM){
    return indiceConv<at::Half>(features, filters, indicePairs, indiceNum, numActOut, _inverse, _subM);
}
std::vector<torch::Tensor> indice_conv_backward_half(torch::Tensor features, torch::Tensor filters,
                 torch::Tensor outGrad, torch::Tensor indicePairs, torch::Tensor indiceNum,
                 int64_t _inverse, int64_t _subM){
    return indiceConvBackward<at::Half>(features, filters, outGrad, indicePairs, indiceNum, _inverse, _subM);
}


torch::Tensor indiceMaxPool_fp_float(torch::Tensor features, torch::Tensor indicePairs, torch::Tensor indiceNum, int64_t numAct){
    return indiceMaxPool<float>(features, indicePairs, indiceNum, numAct);
}
torch::Tensor indiceMaxPool_bp_float(torch::Tensor features, torch::Tensor outFeatures, torch::Tensor outGrad, torch::Tensor indicePairs, torch::Tensor indiceNum){
    return indiceMaxPoolBackward<float>(features, outFeatures, outGrad, indicePairs, indiceNum);
}
torch::Tensor indiceMaxPool_fp_half(torch::Tensor features, torch::Tensor indicePairs, torch::Tensor indiceNum, int64_t numAct){
    indiceMaxPool<at::Half>(features, indicePairs, indiceNum, numAct);
}
torch::Tensor indiceMaxPool_bp_half(torch::Tensor features, torch::Tensor outFeatures, torch::Tensor outGrad, torch::Tensor indicePairs, torch::Tensor indiceNum){
    indiceMaxPoolBackward<at::Half>(features, outFeatures, outGrad, indicePairs, indiceNum);
}
torch::Tensor indiceFieldMaxPool_fp_float(torch::Tensor features, torch::Tensor indicePairs, torch::Tensor indiceNum, int64_t numAct, torch::Tensor feature_norms){
    return indiceFieldMaxPool<float>(features, indicePairs, indiceNum, numAct, feature_norms);
}
torch::Tensor indiceFieldMaxPool_fp_half(torch::Tensor features, torch::Tensor indicePairs, torch::Tensor indiceNum, int64_t numAct, torch::Tensor feature_norms){
    return indiceFieldMaxPool<half>(features, indicePairs, indiceNum, numAct, feature_norms);
}
torch::Tensor indiceAvgPool_fp_float(torch::Tensor features, torch::Tensor indicePairs, torch::Tensor indiceNum, int64_t numAct, torch::Tensor summaryrf){
    return indiceAvgPool<float>(features, indicePairs, indiceNum, numAct, summaryrf);
}
torch::Tensor indiceAvgPool_bp_float(torch::Tensor features, torch::Tensor outFeatures, torch::Tensor outGrad, torch::Tensor indicePairs, torch::Tensor indiceNum, torch::Tensor summaryrf){
    return indiceAvgPoolBackward<float>(features, outFeatures, outGrad, indicePairs, indiceNum, summaryrf);
}
torch::Tensor indiceAvgPool_fp_half(torch::Tensor features, torch::Tensor indicePairs, torch::Tensor indiceNum, int64_t numAct, torch::Tensor summaryrf){
    return indiceAvgPool<at::Half>(features, indicePairs, indiceNum, numAct, summaryrf);
}
torch::Tensor indiceAvgPool_bp_half(torch::Tensor features, torch::Tensor outFeatures, torch::Tensor outGrad, torch::Tensor indicePairs, torch::Tensor indiceNum, torch::Tensor summaryrf){
    return indiceAvgPoolBackward<at::Half>(features, outFeatures, outGrad, indicePairs, indiceNum, summaryrf);
}
torch::Tensor indiceSummaryRF(torch::Tensor indicePairs,
                              torch::Tensor indiceNum, 
                              int64_t numAct){
  auto device             = indicePairs.device().type();
  auto kernelVolume       = indicePairs.size(0);
  auto indicePairNumCpu   = indiceNum.to({torch::kCPU});
  auto options            = torch::TensorOptions().dtype(indicePairs.dtype()).device(indicePairs.device());
  torch::Tensor summarRFs = torch::zeros({numAct}, options);
  for(int i = 0; i<kernelVolume; ++i){
    auto nHot = indicePairNumCpu.data<int>()[i];
    if (nHot <= 0) {
      continue;
    }
    if  (device == torch::kCPU) {
      continue;
    }
    else{
      functor::SummaryRFForwardFunctor<tv::GPU, int> forwardFtor;
      forwardFtor(
        tv::TorchGPU(),
        tv::torch2tv<const int>(indicePairs).subview(i),
        tv::torch2tv<int>(summarRFs),
        nHot
      );
      // printf("summarRFs[0]: %d \n", tv::torch2tv<int>(summarRFs)[0]);
    TV_CHECK_CUDA_ERR();
    }
  }
  return summarRFs;
}

void voxelize_idx_3d(/* long N*4 */ at::Tensor coords, /* long M*4 */ at::Tensor output_coords,
                  /* Int N */ at::Tensor input_map, /* Int M*(maxActive+1) */ at::Tensor output_map, Int batchSize, Int mode){
    voxelize_idx<3>(coords, output_coords, input_map, output_map, batchSize, mode);
}
void voxelize_fp_feat(/* cuda float N*C */ at::Tensor feats, // N * 3 -> M * 3 (N >= M)
              /* cuda float M*C */ at::Tensor output_feats,
              /* cuda Int M*(maxActive+1) */ at::Tensor output_map, Int mode, Int nActive, Int maxActive, Int nPlane){
    voxelize_fp<float>(feats, output_feats, output_map, mode, nActive, maxActive, nPlane);
}
void voxelize_bp_feat(/* cuda float M*C */ at::Tensor d_output_feats, /* cuda float N*C */ at::Tensor d_feats, /* cuda Int M*(maxActive+1) */ at::Tensor output_map,
            Int mode, Int nActive, Int maxActive, Int nPlane){
    voxelize_bp<float>(d_output_feats, d_feats, output_map, mode, nActive, maxActive, nPlane);
}
void point_recover_fp_feat(/* cuda float M*C */ at::Tensor feats, /* cuda float N*C */ at::Tensor output_feats, /* cuda Int M*(maxActive+1) */ at::Tensor idx_map,
                Int nActive, Int maxActive, Int nPlane){
    point_recover_fp<float>(feats, output_feats, idx_map, nActive, maxActive, nPlane);
}
void point_recover_bp_feat(/* cuda float N*C */ at::Tensor d_output_feats, /* cuda float M*C */ at::Tensor d_feats,  /* cuda Int M*(maxActive+1) */ at::Tensor idx_map,
                Int nActive, Int maxActive, Int nPlane){
    point_recover_bp<float>(d_output_feats, d_feats, idx_map, nActive, maxActive, nPlane);
}

